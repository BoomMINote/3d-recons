
#include <hip/hip_runtime.h>
#include <iostream>

__global__ void VecAdd(float* A, float* B, float* C, int N)
{
    int i = threadIdx.x;
	// printf("Thread ID is %d\n", i);
    if (i < N)
        C[i] = A[i] + B[i] + i*4 - i/5 + i*i/1000;
}

int main()
{
    int N = 10000000;
    size_t size = N * sizeof(float);

    // Allocate host memory
    float* h_A = new float[N];
    float* h_B = new float[N];
    float* h_C = new float[N];

    // Initialize host arrays
    for (int i = 0; i < N; i++)
    {
        h_A[i] = static_cast<float>(i);
        h_B[i] = static_cast<float>(2 * i);
    }


    // Allocate device memory
    float* d_A;
    float* d_B;
    float* d_C;
    hipMalloc(&d_A, size);
    hipMalloc(&d_B, size);
    hipMalloc(&d_C, size);

    // Copy host arrays to device
    hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);

    // Kernel invocation with N threads
    VecAdd<<<1000, 10000>>>(d_A, d_B, d_C, N);

    // Copy the result back to host
    hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);

    // Print the result
    for (int i = 0; i < N; i++)
    {
        std::cout << h_C[i] << " ";
    }
    std::cout << std::endl;

    // Free device memory
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    // Free host memory
    delete[] h_A;
    delete[] h_B;
    delete[] h_C;

    return 0;
}